#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres
rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    float t;
    float3 surfNormal;

    Sphere sphere = spheres[primIndex];
    float3 center = sphere.center;
    float radius = sphere.radius;

    // Geometry associated with the sphere
    Matrix<4, 4> transform = sphere.invTransform;
    
    float4 oriTrans = transform * make_float4(ray.origin, 1);
    float4 diTrans = transform * make_float4(ray.direction, 0);
    
    float3 origin = make_float3(oriTrans) / oriTrans.w;
    float3 direction = make_float3(diTrans);

    float a = dot(direction, direction);
    float b = 2 * dot(direction, origin - center);
    float c = dot(origin - center, origin - center) - (radius * radius);
    float deter = b*b - 4*a*c;

    if(deter <= 0)
    {
        t = -1;
    }        
    else
    {
        float x1 = (-b + sqrt(deter)) / (2*a);
        float x2 = (-b - sqrt(deter)) / (2*a);

        if (x2 > 0)
        {
            // outside intersection
            t = x2;
            surfNormal = normalize(make_float3(transform.transpose() * 
            make_float4(origin + t * direction - center, 0)));
        }            
        else        
        {   
            // inside intersection
            t = x1;
            surfNormal = -normalize(make_float3(transform.transpose() * 
            make_float4(origin + t * direction - center, 0)));            
        }            
    }
        
    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // compute surface normal
        attrib.surfNormal = surfNormal;

        // assign material property
        attrib.ambient = sphere.ambient;
        attrib.diffuse = sphere.diffuse;
        attrib.specular = sphere.specular;
        attrib.emission = sphere.emission;
        attrib.shininess = sphere.shininess;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // Sphere bouding box program not implemented for now
    // No acceleration structure is used
    float MIN = -10000.0f;
    float MAX = +10000.0f;

    result[0] = MIN; result[1] = MIN; result[2] = MIN;
    result[3] = MAX; result[4] = MAX; result[5] = MAX;
}