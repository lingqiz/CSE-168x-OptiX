#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

rtBuffer<AreaLight> lights;
rtDeclareVariable(int, maxDepth, , );

// Declare variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 phongBRDF(const float3& kd, const float3& ks, 
    const float s, const float3& lightDir, const float3& reflectDir)
{
    float cosTerm  = max(dot(reflectDir, lightDir), 0.0f);
    float3 lambert = kd / M_PIf;

    float zeroDelta = 0.00001f;
    if (cosTerm < zeroDelta)
        return lambert;

    float3 specular = ks * (s + 2.0f) / (2.0f * M_PIf) * pow(cosTerm, s);
    return lambert + specular;    
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;    
    unsigned int seed = tea<16>(payload.seed, payload.depth);

    // Next Event estimation of the rendering equation
    // Terminate if we hit the light source
    // Return emission for the first bounce
    if (attrib.lightSource)
    {
        payload.recurs = false;
        if(payload.depth == 0)
        {
            payload.radiance = attrib.emission;
        }
            
    }    
    // Otherwise, keep sampling new path through the scene    
    else
    {
        float3 hitPoint = ray.origin + t * ray.direction;
        float3 reflectDir = normalize(ray.direction - 2 * dot(ray.direction, attrib.surfNormal) * attrib.surfNormal);

        // First accumulate direct lighting
        float3 radianceDirect = make_float3(0.0f, 0.0f, 0.0f);
        for(int i = 0; i < lights.size(); i++)
        {
            AreaLight light = lights[i];
            
            float3 radianceSum = make_float3(0.f, 0.f, 0.f);
            float3 lightNormal = normalize(cross(light.ab, light.ac));
            float  lightArea   = length(cross(light.ab, light.ac));
                            
            // Monte Carlo simulation
            int nSample = 36;
            bool stratify = true;
            for(int n = 0; n < nSample; n++)
            {   
                float u = rnd(seed);
                float v = rnd(seed);
                float3 lightLoc;

                if (stratify)
                {
                    int gridSize = (int) sqrt((float) nSample);
                    int x = n / gridSize;
                    int y = n % gridSize;
                    
                    lightLoc = light.a 
                        + ((float) x + u) / (float) gridSize * light.ab
                        + ((float) y + v) / (float) gridSize * light.ac;
                }
                else
                {
                    lightLoc = light.a + u * light.ab + v * light.ac;
                }
                                
                float3 lightDir = normalize(lightLoc - hitPoint);
                float lightDist = length(lightLoc - hitPoint);

                // Light source visibility
                Ray shadowRay = 
                    make_Ray(hitPoint, lightDir, shadowRayIndex, T_MIN, lightDist - T_MIN);
                ShadowPayload shadowPayload;
                shadowPayload.isVisible = true;
                
                rtTrace(root, shadowRay, shadowPayload);
                if(shadowPayload.isVisible)
                {
                    radianceSum +=
                    phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir) * 
                    max(dot(attrib.surfNormal, lightDir), 0.0f) * 
                    max(dot(lightNormal, lightDir), 0.0f) / (lightDist * lightDist);
                }
            }

            radianceDirect += light.col * lightArea / ((float) nSample) * radianceSum;
        }

        payload.radiance += payload.weight * radianceDirect;
        
        // Terminte using a Russian Roulette procedure
        float q = 1 - fminf(fmaxf(payload.weight), 1.0f);
        if (rnd(seed) < q)
        {
            payload.recurs = false;            
        }
        else
        {
            payload.weight /= (1 - q);            
            
            // sample the upper half hemisphere for light ray        
            float3 lightDir = make_float3(0.0f, 0.0f, 0.0f);
            do
            {
                lightDir.x = rnd(seed) * 2.0f - 1.0f;
                lightDir.y = rnd(seed) * 2.0f - 1.0f;
                lightDir.z = rnd(seed) * 2.0f - 1.0f;
            }
            while (length(lightDir) > 1.0f);
            lightDir = normalize(lightDir);

            if(dot(attrib.surfNormal, lightDir) < 0)
                lightDir = -lightDir;

            payload.weight *= (2 * M_PIf) * dot(attrib.surfNormal, lightDir) * 
                phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir);

            payload.origin = hitPoint;
            payload.direction = lightDir;
            payload.depth += 1;
        }
    }        
        
}