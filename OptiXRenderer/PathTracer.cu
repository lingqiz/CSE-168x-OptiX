#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

rtDeclareVariable(int, maxDepth, , );

// Declare variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 phongBRDF(const float3& kd, const float3& ks, 
    const float s, const float3& lightDir, const float3& reflectDir)
{
    float cosTerm  = max(dot(reflectDir, lightDir), 0.0f);
    float3 lambert = kd / M_PIf;

    float zeroDelta = 0.00001f;
    if (cosTerm < zeroDelta)
        return lambert;

    float3 specular = ks * (s + 2.0f) / (2.0f * M_PIf) * pow(cosTerm, s);
    return lambert + specular;    
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;    
    unsigned int seed = tea<16>(payload.seed, payload.depth);

    // Naïve Monte Carlo estimation of the rendering equation
    // Terminate if we hit the light source
    if (attrib.lightSource)
    {
        payload.recurs = false;
        payload.radiance = attrib.emission;
    }    
    // Otherwise, keep sampling new path through the scene
    // Terminte using a Russian Roulette procedure
    else
    {           
        float q = 1 - fminf(fmaxf(payload.weight), 1.0f);
        if (rnd(seed) < q)
        {
            payload.recurs = false;
            payload.radiance = attrib.emission;
        }
        else
        {
            payload.weight /= (1 - q);
            float3 hitPoint = ray.origin + t * ray.direction;
            float3 reflectDir = normalize(ray.direction - 2 * dot(ray.direction, attrib.surfNormal) * attrib.surfNormal);
            
            // sample the upper half hemisphere for light ray        
            float3 lightDir = make_float3(0.0f, 0.0f, 0.0f);
            do
            {
                lightDir.x = rnd(seed) * 2.0f - 1.0f;
                lightDir.y = rnd(seed) * 2.0f - 1.0f;
                lightDir.z = rnd(seed) * 2.0f - 1.0f;
            }
            while (length(lightDir) > 1.0f);
            lightDir = normalize(lightDir);

            if(dot(attrib.surfNormal, lightDir) < 0)
                lightDir = -lightDir;

            payload.weight *= (2 * M_PIf) * dot(attrib.surfNormal, lightDir) * 
                phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir);

            payload.origin = hitPoint;
            payload.direction = lightDir;
            payload.depth += 1;
        }        
    }        
        
}