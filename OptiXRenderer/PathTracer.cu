#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

rtBuffer<AreaLight> lights;
rtDeclareVariable(int, maxDepth, , );

// Declare variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

const float T_MIN = 0.001f;
const int shadowRayIndex = 1;

enum Sampler {uniform, cosine, brdf};

// Compute modified Phong BRDF
static __device__ __inline__ float3 phongBRDF(const float3& kd, const float3& ks, 
    const float s, const float3& lightDir, const float3& reflectDir)
{
    float cosTerm  = max(dot(reflectDir, lightDir), 0.0f);
    float3 lambert = kd / M_PIf;

    float zeroDelta = 0.00001f;
    if (cosTerm < zeroDelta)
        return lambert;

    float3 specular = ks * (s + 2.0f) / (2.0f * M_PIf) * powf(cosTerm, s);
    return lambert + specular;    
}

// PDF approximation of the Phong BRDF function
static __device__ __inline__ float brdfPDF(const float3& surfNormal, const float3& reflectDir, const float3& lightDir, float t, float s)
{
    float diffuse = (1 - t) * max(dot(surfNormal, lightDir), 0.0f) / M_PIf;
    float l_dot_r = dot(reflectDir, lightDir);
    float specular = (l_dot_r <= 0)? 0.0f : (t * (s + 1) / (2 * M_PIf) * pow(dot(reflectDir, lightDir), s));

    return diffuse + specular;
}

// Compute radiance from direct area lighting
static __device__ __inline__ float3 directLight(unsigned int seed, 
    const float3& hitPoint, const float3& reflectDir)
{
    float3 radianceDirect = make_float3(0.0f, 0.0f, 0.0f);
    for(int i = 0; i < lights.size(); i++)
    {
        AreaLight light = lights[i];
        
        float3 radianceSum = make_float3(0.f, 0.f, 0.f);
        float3 lightNormal = normalize(cross(light.ab, light.ac));
        float  lightArea   = length(cross(light.ab, light.ac));
                        
        // Monte Carlo integration of direct lighting
        int nSample = 9;        
        for(int n = 0; n < nSample; n++)
        {   
            // Stratified sampling for area light source
            float u = rnd(seed);
            float v = rnd(seed);
            float3 lightLoc;
            
            int gridSize = (int) sqrt((float) nSample);
            int x = n / gridSize;
            int y = n % gridSize;
            
            lightLoc = light.a 
                + ((float) x + u) / (float) gridSize * light.ab
                + ((float) y + v) / (float) gridSize * light.ac;
                                        
            float3 lightDir = normalize(lightLoc - hitPoint);
            float lightDist = length(lightLoc - hitPoint);

            // Light source visibility
            Ray shadowRay = 
                make_Ray(hitPoint, lightDir, shadowRayIndex, T_MIN, lightDist - T_MIN);
            ShadowPayload shadowPayload;
            shadowPayload.isVisible = true;
            
            rtTrace(root, shadowRay, shadowPayload);
            if(shadowPayload.isVisible)
            {
                radianceSum +=
                phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir) * 
                max(dot(attrib.surfNormal, lightDir), 0.0f) * 
                max(dot(lightNormal, lightDir), 0.0f) / (lightDist * lightDist);
            }
        }

        radianceDirect += light.col * lightArea / ((float) nSample) * radianceSum;
    }

    return radianceDirect;
}

// Uniformlly sample the upper hemisphere
static __device__ __inline__ float3 uniformSampler(unsigned int seed, const float3& surfNormal)
{
    float3 lightDir = make_float3(0.0f, 0.0f, 0.0f);
    do
    {
        lightDir.x = rnd(seed) * 2.0f - 1.0f;
        lightDir.y = rnd(seed) * 2.0f - 1.0f;
        lightDir.z = rnd(seed) * 2.0f - 1.0f;
    }
    while (length(lightDir) > 1.0f);
    lightDir = normalize(lightDir);

    if(dot(surfNormal, lightDir) < 0)
        return -lightDir;

    return lightDir;
}

// Sample with cosine PDF
static __device__ __inline__ float3 cosSampler(unsigned int seed, const float3& surfNormal)
{
    float theta = acosf(sqrtf(rnd(seed)));
    float phi = 2 * M_PIf * rnd(seed);

    float3 s = make_float3(cosf(phi) * sinf(theta), 
        sinf(phi) * sinf(theta), cosf(theta));

    float3 a = make_float3(1.0f, 0.f, 0.f);
    if(1 - dot(a, surfNormal) < 0.1f)
        a = make_float3(0.f, 1.0f, 0.f);

    float3 u = normalize(cross(a, surfNormal));
    float3 v = cross(surfNormal, u);

    return s.x * u + s.y * v + s.z * surfNormal;
}

// Sample from PDF adapted from Phong BRDF
static __device__ __inline__ float3 brdfSampler(unsigned int seed, const float3& surfNormal, const float3& reflectDir, float t, float s)
{    
    if(rnd(seed) <= t)
    {
        // sample the specular component
        float theta = acosf(powf(rnd(seed), 1.0f / (s + 1.0f)));
        float phi = 2 * M_PIf * rnd(seed);

        float3 s = make_float3(cosf(phi) * sinf(theta), 
            sinf(phi) * sinf(theta), cosf(theta));

        float3 a = make_float3(1.0f, 0.f, 0.f);
        if(1 - dot(a, reflectDir) < 0.1f)
            a = make_float3(0.f, 1.0f, 0.f);

        float3 u = normalize(cross(a, reflectDir));
        float3 v = cross(reflectDir, u);
        
        return s.x * u + s.y * v + s.z * reflectDir;
    }

    // sample the diffuse component
    return cosSampler(seed, surfNormal);
}

// Main path tracing routine
RT_PROGRAM void closestHit()
{    
    unsigned int seed = tea<16>(payload.seed, payload.depth);

    // Next event estimation of the rendering equation
    // Terminate if we hit the light source
    // Return emission for the first bounce
    if (attrib.lightSource)
    {
        payload.recurs = false;
        if(payload.depth == 0)        
            payload.radiance = attrib.emission;
    }             
    else
    {
        float3 hitPoint = ray.origin + t * ray.direction;
        float3 reflectDir = 
            normalize(ray.direction - 2 * dot(ray.direction, attrib.surfNormal) * attrib.surfNormal);

        // Otherwise, accumulate the emission and direct lighting term first
        payload.radiance += 
            payload.weight * (attrib.emission + directLight(seed, hitPoint, reflectDir));
                
        // Terminte using a Russian Roulette procedure
        float q = 1 - fminf(fmaxf(payload.weight), 1.0f);
        if (rnd(seed) < q)
        {
            payload.recurs = false;
        }
        else
        {
            // Reweight path contribution            
            payload.weight /= (1 - q);

            // Sample next indirect path
            // and update the contribution of the new path
            float3 lightDir;

            Sampler sampler = brdf;
            switch (sampler)
            {
                case uniform:
                    lightDir = uniformSampler(seed, attrib.surfNormal);
                    payload.weight *= (2 * M_PIf) * dot(attrib.surfNormal, lightDir) * 
                    phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir);
                break;

                case cosine:
                    lightDir = cosSampler(seed, attrib.surfNormal);
                    payload.weight *= M_PIf * 
                    phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir);
                break;

                case brdf:
                    lightDir = brdfSampler(seed, attrib.surfNormal, reflectDir, attrib.brdf_t, attrib.shininess);
                    if(dot(lightDir, attrib.surfNormal) <= 0)
                    {                        
                        payload.weight *= make_float3(0.0f, 0.0f, 0.0f);
                    }
                    else
                    {
                        payload.weight *= dot(attrib.surfNormal, lightDir) * 
                        phongBRDF(attrib.diffuse, attrib.specular, attrib.shininess, lightDir, reflectDir) /
                        brdfPDF(attrib.surfNormal, reflectDir, lightDir, attrib.brdf_t, attrib.shininess);
                    }                    
                break;
            }

            // Return and trace the new path
            payload.origin = hitPoint;
            payload.direction = lightDir;
            payload.depth += 1;
        }
    }   
}