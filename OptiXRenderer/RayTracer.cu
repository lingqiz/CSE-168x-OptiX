#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers and variable
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtDeclareVariable(float3, attenu, , );
rtDeclareVariable(int, maxDepth, , );

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 
computeShading(const float3& lightDir, const float3& lightColor, const float3& normalVector,
const float3& halfVector, const float3& diffuse, const float3& specular, const float shininess)
{
    float n_dot_l = max(dot(normalVector, lightDir), 0.0f);
    float3 lambert = diffuse * lightColor * n_dot_l;

    float n_dot_h = max(dot(normalVector, halfVector), 0.0f);
    float3 phong   = specular * lightColor * pow(n_dot_h, shininess);

    return lambert + phong;
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;

    float3 radiance = attrib.ambient + attrib.emission;
    float3 hitPoint = ray.origin + t * ray.direction;

    // compute shading for point light
    for(int i = 0; i < plights.size(); i++)
    {   
        PointLight light = plights[i];
        float3 lightDir  = normalize(light.loc - hitPoint);
        float lightDist = length(light.loc - hitPoint);

        // Cast shadow ray and test for light source visibility
        Ray shadowRay = make_Ray(hitPoint, lightDir, shadowRayIndex, T_MIN, lightDist);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;

        rtTrace(root, shadowRay, shadowPayload);

        if(shadowPayload.isVisible)
        {
            float3 halfVector = normalize(lightDir - ray.direction);
            float attenuConst = attenu.x + attenu.y * lightDist + attenu.z * (lightDist * lightDist);
            
            radiance += computeShading(lightDir, light.col / attenuConst, attrib.surfNormal, 
                halfVector, attrib.diffuse, attrib.specular, attrib.shininess);
        }
    }

    // compute shading for direct light
    for(int i = 0; i < dlights.size(); i++)
    {
        DirectionalLight light = dlights[i];
        float3 lightDir = normalize(light.loc);

        // Cast shadow ray and test for light source visibility
        Ray shadowRay = make_Ray(hitPoint, lightDir, shadowRayIndex, T_MIN, RT_DEFAULT_MAX);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;

        rtTrace(root, shadowRay, shadowPayload);

        if(shadowPayload.isVisible)
        {
            float3 halfVector = normalize(lightDir - ray.direction);
            radiance += computeShading(lightDir, light.col, attrib.surfNormal, 
                halfVector, attrib.diffuse, attrib.specular, attrib.shininess);
        }
    }
    
    // Set radiance of current ray    
    payload.radiance = payload.specular * radiance;
   
    // recursive trace
    float zeroDelta = 0.001f;
    if(length(attrib.specular) < zeroDelta || payload.depth > maxDepth)
    {
        payload.recurs = false;
    }
    else
    {   
        // payload.recurs = true;
        // light ray for reflection
        payload.origin = hitPoint;
        payload.direction = ray.direction - 2 * dot(ray.direction, attrib.surfNormal) * attrib.surfNormal;
        payload.specular *= attrib.specular;
        payload.depth += 1;
    }
        
}