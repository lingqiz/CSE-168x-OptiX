#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{        
    payload.radiance = attrib.ambient;
}