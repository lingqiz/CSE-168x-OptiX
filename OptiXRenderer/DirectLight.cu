#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers and variable
rtBuffer<AreaLight> lights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 
computeShading(const float3& lightDir, const float3& lightColor, const float3& normalVector,
const float3& halfVector, const float3& diffuse, const float3& specular, const float shininess)
{
    float n_dot_l = max(dot(normalVector, lightDir), 0.0f);
    float3 lambert = diffuse * lightColor * n_dot_l;

    float n_dot_h = max(dot(normalVector, halfVector), 0.0f);
    float3 phong   = specular * lightColor * pow(n_dot_h, shininess);

    return lambert + phong;
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;

    float3 radiance = attrib.ambient + attrib.emission;
    float3 hitPoint = ray.origin + t * ray.direction;

    // Physically based rendering for area lights
   
    // turn off recursive trace
    payload.recurs = false;
        
}