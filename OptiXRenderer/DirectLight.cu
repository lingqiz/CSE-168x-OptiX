#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers and variable
rtBuffer<AreaLight> lights;
rtDeclareVariable(int, nSample, , );
rtDeclareVariable(int, stratify, , );

// Declare variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 phongBRDF(const float3& kd, const float3& ks, 
    const float s, const float3& lightDir, const float3& reflectDir)
{
    float3 lambert = kd / M_PIf;
    float3 specular = ks * (s + 2) / (2 * M_PIf) * pow(dot(reflectDir, lightDir), s);
    return lambert + specular;
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;

    // We slove the first two terms of the rendering equation:
    // emission and direct lighting

    float3 radiance = attrib.emission;
    float3 hitPoint = ray.origin + t * ray.direction;

    // Physically based rendering for area lights
    unsigned int seed = tea<16>(launchIndex.x, launchIndex.y);
    for(int i = 0; i < lights.size(); i++)
    {
        AreaLight light = lights[i];
        float3 radianceSum = make_float3(0.f, 0.f, 0.f);
        float3 lightNormal = normalize(cross(light.ab, light.ac));
        float  lightArea   = length(cross(light.ab, light.ac));        
        
        for(int n = 0; n < nSample; n++)
        {
            float u = rnd(seed);
            float v = rnd(seed);                        
            rtPrintf("%.2f, %.2f \n", u, v);
        }
        radiance += radianceSum / (float) nSample;
    }
   
    // turn off recursive trace
    payload.recurs = false;
        
}