#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers and variable
rtBuffer<AreaLight> lights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// ray and intersection related
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

static __device__ __inline__ float3 phongBRDF(const float3& kd, const float3& ks, 
    const float s, const float3& lightDir, const float3& reflectDir)
{
    float3 lambert = kd / M_PIf;
    float3 specular = ks * (s + 2) / (2 * M_PIf) * pow(dot(reflectDir, lightDir), s);
    return lambert + specular;
}

RT_PROGRAM void closestHit()
{
    const float T_MIN = 0.001f;    
    const int shadowRayIndex = 1;

    float3 radiance = attrib.ambient + attrib.emission;
    float3 hitPoint = ray.origin + t * ray.direction;

    // Physically based rendering for area lights
    
   
    // turn off recursive trace
    payload.recurs = false;
        
}