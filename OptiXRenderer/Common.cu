#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "Payloads.h"

using namespace optix;

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );

RT_PROGRAM void miss()
{
    // Set the result to be the background color if miss
    payload.radiance = backgroundColor;
    payload.done = true;
}

RT_PROGRAM void exception()
{
    // Print any exception for debugging
    const unsigned int code = rtGetExceptionCode();
    rtPrintExceptionDetails();
}

rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );

RT_PROGRAM void anyHit()
{
    shadowPayload.isVisible = false;
    rtTerminateRay();
}