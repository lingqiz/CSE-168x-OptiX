#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)
rtDeclareVariable(int1, frameID, , );

// Camera info 
rtDeclareVariable(float, width, , );
rtDeclareVariable(float, height, , );
rtDeclareVariable(float3, dir, , );
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, camFrom, , );
rtDeclareVariable(float, fovxRad, , );
rtDeclareVariable(float, fovyRad, , );

RT_PROGRAM void generateRays()
{
    // Calculate the ray direction
    // Note that the indices are flipped due to column major convention
    float T_MIN = 0.001f;
    float idw = ((float) launchIndex.x) + 0.5f;
    float idh = ((float) launchIndex.y) + 0.5f;

    float alpha = tan(fovxRad / 2.0f) * (idw - width / 2.0f) / (width / 2.0f);
    float beta  = tan(fovyRad / 2.0f) * (height / 2.0f - idh) / (height / 2.0f);
    float3 rayDir = normalize(alpha * u + beta * v - dir);

    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(camFrom, rayDir, 0, T_MIN, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);
        
    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}