#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)
rtDeclareVariable(int1, frameID, , );

// Camera info 
rtDeclareVariable(float, width, , );
rtDeclareVariable(float, height, , );
rtDeclareVariable(float3, dir, , );
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, camFrom, , );
rtDeclareVariable(float, fovxRad, , );
rtDeclareVariable(float, fovyRad, , );

RT_PROGRAM void generateRays()
{    
    const float T_MIN = 0.0001f;
    const int primRayIndex = 0;

    // Calculate the ray direction
    // x: width variable, y: height variable
    // (0, 0) is at upper left corner
    float idw = ((float) launchIndex.x) + 0.5f;
    float idh = ((float) launchIndex.y) + 0.5f;

    float alpha = tan(fovxRad / 2.0f) * (idw - width / 2.0f) / (width / 2.0f);
    float beta  = tan(fovyRad / 2.0f) * (height / 2.0f - idh) / (height / 2.0f);
    float3 rayDir = normalize(alpha * u + beta * v - dir);

    // Set up variable for recursive ray tracing
    float3 result = make_float3(0.0f, 0.0f, 0.0f);

    Payload payload;
    payload.depth = 0; payload.recurs = true;
    payload.origin = camFrom; payload.direction = rayDir;
    payload.specular = make_float3(1.0f, 1.0f, 1.0f);
    
    do
    {
        Ray ray = make_Ray(payload.origin, payload.direction, primRayIndex, T_MIN, RT_DEFAULT_MAX);
        rtTrace(root, ray, payload);
        
        result += payload.radiance;
    } 
    while(payload.recurs);
            
    // Write the result
    resultBuffer[launchIndex] = result;
}